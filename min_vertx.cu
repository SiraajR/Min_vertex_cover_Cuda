#include "hip/hip_runtime.h"

//#include <stdc++.h>
#include <cstdio>
#include <cstdlib>
#include <hip/hip_runtime.h>
#include <cstring>
#include <iostream>
#include <hip/hip_runtime.h>
#include "un_graph.h"
using namespace std;

#define MAX_THREAD 1024

void catchError(hipError_t error , const char* funct){
    if (error  != hipSuccess){
        printf("\n>>>>>Cuda error code %i <<<<<\n %s line of error and function : %s\n" , error , hipGetErrorString(error) , funct);
        exit(-1);
    }
   
}

__global__ void kernel1(int* Mvc , int* PMvc , int* AdjList , int* AdjListPtr , int* Deg , int num_node){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= num_node){
        return;
    }
    int deg = Deg[tid];
    int mdeg = deg;
    for(int i = AdjListPtr[tid] ; i < AdjListPtr[tid + 1] ; i ++ ){
        int neighbour = AdjList[i];
        int deg2 = Deg[neighbour];
        mdeg = min(mdeg , deg2);
    }
    if (deg == mdeg){
        Mvc[tid] = 0;

    }
}
__global__ void kernel2(int* Mvc , int * PMvc , int* AdjList , int* AdjListPtr , int* Adj , int* terminate , int num_node){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= num_node){
        return;
    }
    Adj[tid] = 1;
    for(int i = AdjListPtr[tid] ; i < AdjListPtr[tid + 1] ; i++){
        int neighbour = AdjList[i];
        if(Mvc[neighbour] == 0){
            Adj[tid] = 0;
            break;
        }
    }
    //printf("\n🧵 Thread ID: %d\n", tid);
    //printf("╭──────────────────────────────╮\n");
    //printf("│ Mvc[tid]  = %d\n" ,  Mvc[tid]);
    //printf("│ PMvc[tid] = %d\n", PMvc[tid]);
    //printf("│ Adj[tid]  = %d\n", Adj[tid]);
    //printf("╰──────────────────────────────╯\n");

    if (Mvc[tid] != (1 - Adj[tid])) {
        *terminate = 0;
        //printf("🔁 Recompute Triggered by tid %d ⚠️\n", tid);
    } else {
        //printf("✅ Computation Complete for tid %d\n", tid);
    }
    PMvc[tid] = Mvc[tid];
}
__global__ void kernel3(int* Mvc , int * PMvc , int* AdjList , int* AdjListPtr , int* Adj , int num_node){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= num_node){
        return;
    }
    bool u_cond = false;
    bool w_cond = false;    
    for(int i = AdjListPtr[tid] ; i < AdjListPtr[tid + 1] ; i++){
        int neighbour = AdjList[i];
        if((Mvc[neighbour] == 1 &&  Adj[neighbour] == 0 && Adj[tid] == 1 )){
            u_cond = true;
            
        }
        if(Mvc[neighbour] == 0 && Adj[neighbour] == 1){
            w_cond = true;
        }
        if(u_cond && w_cond){
            break;
        }
    }  

    if(u_cond && !w_cond){
        Mvc[tid] = 0;
    } 
}
__global__ void kernel4(int* PMvc , int* AdjListPtr , int* Adj , int* AdjList, int* Mvc , int num_node ){
    int tid = blockDim.x * blockIdx.x + threadIdx.x;
    if(tid >= num_node){
        return;
    }
    if (PMvc[tid] == 0 && Adj[tid] == 0){
        for (int i = AdjListPtr[tid] ; i < AdjListPtr[tid + 1] ; i++){
            int neighbour = AdjList[i];
            if(PMvc[neighbour] == 0 && Adj[neighbour] == 0 && neighbour < tid){
                Mvc[tid] = 1;

            }

        }
    }
}
int main(int argc, char *argv[]){
    if(argc < 2){
        cout << "Usage:" << argv[0] << "<graph_input> [output_file] \n";
        return 0;
    }
    if (!freopen(argv[1] , "r" , stdin)){
        perror("Failed to open the file");
        return 1;
    }
    
    
    int* D_AdjList , *D_AdjListPtr , *D_Deg;
    int* d_Mvc , * d_Adj , *d_PMvc;
    //Cuda Event creating 
    hipEvent_t start , stop;

    catchError(hipEventCreate(&start) , "Event Creation start");
    catchError(hipEventCreate(&stop) , "Event Creation stop");
    un_graph *host_graph = new un_graph();
    un_graph *device_Graph;
    catchError(hipMalloc((void**) &device_Graph , sizeof(un_graph)) , "Mem alloc for graph");
    host_graph->graph();
    int num_node = host_graph -> getNode();
    int num_edge = host_graph-> getEdges();
    int* h_Mvc;
    int* h_Adj ;
    int* h_PMvc ;
    // minimum vertex cover
    int* V;
    catchError(hipHostMalloc((void**) &V , num_node * sizeof(int)) , "Min vertex cover Mem Alloc");

    int* H_AdjList = host_graph->adjList;
    int* H_AdjListPtr = host_graph->adjListPtr;
    int* H_Deg= host_graph->Degr;
    int* D_terminate;
    //Allocate memory to memcpy 
    catchError(hipHostMalloc((void**)&h_Mvc , num_node*sizeof(int)) , "host side MVC alloc");
    catchError(hipHostMalloc((void**)&h_Adj , num_node*sizeof(int)) , "host side Adj mem alloc");
    catchError(hipHostMalloc((void**)&h_PMvc , num_node*sizeof(int)) , "Previous PMvc host side malloc ");

    //initializing the host arrays with dummy values 
    for (int i = 0; i < num_node; ++i) h_Mvc[i] = 1;
    memset(h_Adj , 0 , num_node * sizeof(int));
   // memset(h_PMvc , 0 , num_node * sizeof(int));

    //Allocating memory
    catchError(hipMalloc((void**)&d_Mvc, num_node * sizeof(int)) , "device side MVC Malloc");
    catchError(hipMalloc((void**)&d_Adj, num_node * sizeof(int)) , "device side Adj Malloc");
    catchError(hipMalloc((void**)&d_PMvc, num_node * sizeof(int)) , "device side PMVC Malloc");
    catchError(hipMalloc((void**)&D_AdjList, (2*num_edge + 1) * sizeof(int)) , "device side AdjList Malloc");
    catchError(hipMalloc((void**)&D_AdjListPtr, (num_node + 1) * sizeof(int)) , "device side AdjListPtr Malloc");
    catchError(hipMalloc((void**)&D_Deg, num_node * sizeof(int)) , "device side Deg Malloc");
    catchError(hipMalloc((void**) &D_terminate , sizeof(int)) , "device side terminate Malloc");

    
    catchError(hipMemcpy(D_AdjList , H_AdjList , sizeof(int)* (2*num_edge + 1) , hipMemcpyHostToDevice) , "MemCpy1");
    catchError(hipMemcpy(D_AdjListPtr , H_AdjListPtr , sizeof(int) * (num_node + 1) , hipMemcpyHostToDevice) , "MemCpy2");
    catchError(hipMemcpy(D_Deg , H_Deg , num_node*sizeof(int) , hipMemcpyHostToDevice) , "MemCpy3");
    catchError(hipMemcpy(device_Graph , host_graph , sizeof(un_graph) , hipMemcpyHostToDevice) , "MemCpy4");
    catchError(hipMemcpy(d_Mvc , h_Mvc, num_node * sizeof(int) , hipMemcpyHostToDevice) , "MemCpy5");
    catchError(hipMemcpy(d_Adj , h_Adj , num_node * sizeof(int) , hipMemcpyHostToDevice) , "MemCpy6");
    catchError(hipMemcpy(d_PMvc ,h_PMvc , num_node*sizeof(int) , hipMemcpyHostToDevice) , "MemCpy7");

   

    int Num_blocks = (num_node + MAX_THREAD - 1) / MAX_THREAD;
    catchError(hipEventRecord(start) , "start event record");
    kernel1<<<Num_blocks  , MAX_THREAD>>>(d_Mvc, d_PMvc,  D_AdjList,  D_AdjListPtr, D_Deg , num_node);
    catchError(hipGetLastError() , "last err");
    catchError(hipMemcpy(d_PMvc , d_Mvc , num_node * sizeof(int) , hipMemcpyDeviceToDevice) , "MemCpy8");
    int H_terminate = 1;

    do{
        H_terminate = 1;
        hipMemcpy(D_terminate , &H_terminate , sizeof(int) , hipMemcpyHostToDevice );
        kernel3<<<Num_blocks , MAX_THREAD>>>(d_Mvc, d_PMvc, D_AdjList, D_AdjListPtr, d_Adj , num_node);
        //kernel 3 initialization
        catchError(hipGetLastError() , "kernel 3");
        kernel2<<<Num_blocks , MAX_THREAD>>>(d_Mvc, d_PMvc, D_AdjList, D_AdjListPtr, d_Adj, D_terminate , num_node);
        //kernel 2 initialization
        catchError(hipGetLastError() ,"kernel 2 pt1");
        kernel4<<<Num_blocks , MAX_THREAD>>>(d_PMvc, D_AdjListPtr, d_Adj, D_AdjList, d_Mvc , num_node);
        //kernel 4 initialization 
        catchError(hipGetLastError() , "kernel 4");
        kernel2<<<Num_blocks , MAX_THREAD>>>(d_Mvc, d_PMvc, D_AdjList, D_AdjListPtr, d_Adj, D_terminate , num_node);
        //syncing PMvc and Mvc
        catchError(hipGetLastError() ,"kernel 2 pt2");
        //hipDeviceSynchronize();
        hipMemcpy(&H_terminate , D_terminate , sizeof(int) , hipMemcpyDeviceToHost);
        //printf("H_terminate = %d\n", H_terminate);
        /*if(H_terminate == 0) {
            printf("🔁 Recompute Triggered by tid");
        }else{
            printf("✅ Computation Complete");
        }
        */
        //hipMemcpy(d_PMvc , d_Mvc , num_node * sizeof(int) , hipMemcpyDeviceToDevice);
        //kernel 2 initialization
    
    }while(H_terminate == 0); // if its 0 then something changed , if its 1 then its over

    printf("Number of Nodes: %d\n" , num_node);
    printf("Number of Edges: %d\n" , num_edge);

    catchError(hipMemcpy(V , d_Mvc , num_node*sizeof(int) , hipMemcpyDeviceToHost) , "MemCpy9");
    catchError(hipEventRecord(stop) , "Event record: stop");
    catchError(hipEventSynchronize(stop) , "Event sync");
    float total_time = 0;
    catchError(hipEventElapsedTime( &total_time, start , stop) , "Total time taken: ");
    printf("GPU time takes: %f\n" , total_time);
    if(argc == 3){
        if (!freopen(argv[2], "w", stdout)) {
            perror("Failed to open output file");
            return 1;
        }
        for(int i = 0 ; i < num_node ; ++i){
            cout << V[i] << " ";
        }
        cout << endl;
    }

    // freeing up allocated memeory
    //cuda free device side memory allocation

    //host side freeing 
    catchError(hipHostFree(h_Mvc) , "free h_Mvc");
    catchError(hipHostFree(h_Adj) , "free h_Adj");
    catchError(hipHostFree(h_PMvc) , "free h_PMvc");
    
    //device side freeing 
    catchError(hipFree(d_Mvc) , "free d_Mvc");
    catchError(hipFree(d_Adj) , "free d_adj");
    catchError(hipFree(d_PMvc) , "free d_PMvc");
    catchError(hipFree(D_AdjList) , "free D_AdjList");
    catchError(hipFree(D_AdjListPtr) , "free D_AdjListPtr");
    catchError(hipFree(D_Deg) , "free D_Deg");
    catchError(hipFree(D_terminate) , "free D_terminate");
}